#include <new>
#include <stdio.h>
#include "mm.cuh"

#define N 10

int main() {
    double a[N * N], b[N * N], c[N * N];
    double *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N * N; ++i) {
        a[i] = 0;
        b[i] = 0;
        c[i] = 0;
    }

    for (int i = 0; i < N; ++i) {
        a[N * i + i] = (i + 1);
        b[N * i + i] = 1 / (double)(i + 1);
    }

    hipMalloc((void **)&dev_a, N * N * sizeof(double));
    hipMalloc((void **)&dev_b, N * N * sizeof(double));
    hipMalloc((void **)&dev_c, N * N * sizeof(double));

    hipMemcpy(dev_a, a, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 grid(N, N);
    dim3 block(1);
    mm<<<grid, block>>>(dev_a, dev_b, dev_c, N, N, N);

    hipMemcpy(c, dev_c, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", a[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", b[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", c[N * i + j]);
        }
        printf("\n");
    }

    return 0;
}
