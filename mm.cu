
#include <hip/hip_runtime.h>
__global__ void mm(double *a, double *b, double *c, int dim_l, int dim_m, int dim_n) {
    int size = dim_l * dim_n;
    int x = blockIdx.x;
    int y = blockIdx.y;

    for (int k = 0; k < dim_n; k++) {
        c[dim_m * x + y] += a[dim_m * x + k] * b[dim_n * k + y];
    }
}
