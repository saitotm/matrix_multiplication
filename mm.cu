
#include <hip/hip_runtime.h>
#include <new>
#include <stdio.h>

#define N 3

__device__ int calc_index(int n, int m, int col_x, int col_y) { return m * col_x + col_y; }

__global__ void mm(double* a, double* b, double* c, int l, int m, int n) {
    int size = l * n;
    for (int i = 0; i < l; ++i) {
        for (int j = 0; j < n; ++j) {
            c[calc_index(l, m, i, j)] = 0;
        }
    }

    for (int i = 0; i < l; i++) {
        for (int j = 0; j < m; j++) {
            for (int k = 0; k < n; k++) {
                c[calc_index(l, m, i, j)] +=
                    a[calc_index(l, m, i, k)] * b[calc_index(m, n, k, j)];
            }
        }
    }
}

int main() {
    double a[N * N], b[N * N], c[N * N];
    double *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N * N; ++i) {
        a[i] = 0;
        b[i] = 0;
    }

    for (int i = 0; i < N; ++i) {
        a[N * i + i] = (i + 1);
        b[N * i + i] = 1 / (double)(i + 1);
    }

    hipMalloc((void **)& dev_a, N * N * sizeof(double));
    hipMalloc((void **)& dev_b, N * N * sizeof(double));
    hipMalloc((void **)& dev_c, N * N * sizeof(double));

    hipMemcpy(dev_a, a, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(1);
    mm<<<grid, block>>>(dev_a, dev_b, dev_c, N, N, N);

    hipMemcpy(c, dev_c, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", a[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", b[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", c[N * i + j]);
        }
        printf("\n");
    }


    return 0;
}
