
#include <hip/hip_runtime.h>
#include <new>
#include <stdio.h>

#define N 3

__global__ void mm(double* a, double* b, double* c, int dim_l, int dim_m, int dim_n) {
    int size = dim_l * dim_n;

    for (int i = 0; i < dim_l; i++) {
        for (int j = 0; j < dim_m; j++) {
            for (int k = 0; k < dim_n; k++) {
                c[dim_m * i + j] +=
                    a[dim_m * i + k] * b[dim_n * k + j];
            }
        }
    }
}

int main() {
    double a[N * N], b[N * N], c[N * N];
    double *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N * N; ++i) {
        a[i] = 0;
        b[i] = 0;
        c[i] = 0;
    }

    for (int i = 0; i < N; ++i) {
        a[N * i + i] = (i + 1);
        b[N * i + i] = 1 / (double)(i + 1);
    }

    hipMalloc((void **)& dev_a, N * N * sizeof(double));
    hipMalloc((void **)& dev_b, N * N * sizeof(double));
    hipMalloc((void **)& dev_c, N * N * sizeof(double));

    hipMemcpy(dev_a, a, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(1);
    mm<<<grid, block>>>(dev_a, dev_b, dev_c, N, N, N);

    hipMemcpy(c, dev_c, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", a[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", b[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", c[N * i + j]);
        }
        printf("\n");
    }


    return 0;
}
