
#include <hip/hip_runtime.h>
#include <new>
#include <stdio.h>

#define N 3

int calc_index(int n, int m, int col_x, int col_y) { return m * col_x + col_y; }

void mm(double* a, double* b, double* c, int l, int m, int n) {
    int size = l * n;

    for (int i = 0; i < l; i++) {
        for (int j = 0; j < m; j++) {
            for (int k = 0; k < n; k++) {
                c[calc_index(l, m, i, j)] +=
                    a[calc_index(l, m, i, k)] * b[calc_index(m, n, k, j)];
            }
        }
    }
}

int main() {
    double a[N * N], b[N * N], c[N * N];
    double *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N * N; ++i) {
        a[i] = 0;
        b[i] = 0;
        c[i] = 0;
    }

    for (int i = 0; i < N; ++i) {
        a[N * i + i] = (i + 1);
        b[N * i + i] = 1 / (double)(i + 1);
    }

    mm(a, b, c, N, N, N);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", a[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", b[N * i + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", c[N * i + j]);
        }
        printf("\n");
    }

    return 0;
}
